#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <culapackdevice.h>

void checkStatus(culaStatus status)
{
    char buf[80];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    //culaShutdown();
    //exit(EXIT_FAILURE);
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    //culaShutdown();
    //exit(EXIT_FAILURE);
}

void checkCublasStatus(hipblasStatus_t stat)
{
	if (stat != HIPBLAS_STATUS_SUCCESS ) {
		printf ( "CUBLAS Error \n" );

		//culaShutdown();
		//exit(EXIT_FAILURE);
	}	    
}

#define TILE_DIM    16
#define BLOCK_ROWS  16

// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transpose(float *odata, float* idata, int width, int height)
{
	int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
  
	int index  = xIndex + width*yIndex;
  
	for (int i=0; i<TILE_DIM; i+=16) {
		odata[index+i*width] = idata[index+i*width];
	}  
}

__global__ void vector_add(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)
        C[i] = A[i] + B[i];
}

__global__ void calc_r(float* pi, float* u, float* r, float* vd, float v2, int l, int m){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	int ld = l-1;

	if (i < m){
		pi[i] = u[ld + i*l];
	    v2 += powf(pi[i], 2);
	}

	if (j < ld){
		r[j] += vd[j + i*ld] * pi[i];
	}
}


void vssa(int n, int l, int p, int* pp, int m, float* timeseries, float* forecast, int count){
	hipError_t err;
    culaStatus status;
	hipblasStatus_t stat ;
	hipblasHandle_t handle ;

	int f_size = n + m + l - 1; 
	int k = n - l + 1;
	int ld = l - 1;
	
	//Init and copy timeseries to device
	float* d_timeseries;

	err = hipMalloc(&d_timeseries, (n+count)*sizeof(float));
	checkCudaError(err);

	err = hipMemcpy(d_timeseries, timeseries, (n+count)*sizeof(float), hipMemcpyHostToDevice);
	checkCudaError(err);

	//Init and copy forecast 
	float* d_forecast;
	
	err = hipMalloc(&d_forecast, f_size*count*sizeof(float));
	checkCudaError(err);

	//Init Cublas
	stat = hipblasCreate(&handle);
	checkCublasStatus(stat);

	//Init CULA
	status = culaInitialize();
    checkStatus(status);
	
	//Init local variable
	float* x;
	err = hipMalloc(&x, l*k*sizeof(float));
	checkCudaError(err);

	float *s;
	err = hipMalloc(&s, l*sizeof(float));
	checkCudaError(err);

	float *u;
	err = hipMalloc(&u, l*l*sizeof(float));
	checkCudaError(err);

	float *vt;
	err = hipMalloc(&vt, k*k*sizeof(float));
	checkCudaError(err);

	float *v;
	err = hipMalloc(&v, k*k*sizeof(float));
	checkCudaError(err);

	float *xi;
	err = hipMalloc(&xi, l*k*sizeof(float));
	checkCudaError(err);

	float *ui;
	err = hipMalloc(&ui, l*sizeof(float));
	checkCudaError(err);

	float *vi;
	err = hipMalloc(&vi, k*sizeof(float));
	checkCudaError(err);

	float *xii;
	err = hipMalloc(&xii, l*sizeof(float));
	checkCudaError(err);

	float *pi;
	err = hipMalloc(&pi, m*sizeof(float));
	checkCudaError(err);

	float *vd;
	err = hipMalloc(&vd, ld*m*sizeof(float));
	checkCudaError(err);

	float *v2;
	err = hipMalloc(&v2, sizeof(float));
	checkCudaError(err);

	float *r;
	err = hipMalloc(&r, ld*sizeof(float));
	checkCudaError(err);

	float *vdxvdt;
	err = hipMalloc(&vdxvdt, ld*ld*sizeof(float));
	checkCudaError(err);

	float *rxrt;
	err = hipMalloc(&rxrt, ld*ld*sizeof(float));
	checkCudaError(err);
	
	float *pr;
	err = hipMalloc(&pr, ld*ld*sizeof(float));
	checkCudaError(err);

	//Execute vssa for count points
	for (int index = 0; index < count; ++index){
		//Populate trajectory matrix
		for (int j = 0; j < k; ++j){
			err = hipMemcpy(x + j*l, d_timeseries + (j + index), l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
        }

		//Execute SVD
		status = culaDeviceSgesvd('S', 'S', l, k, x, l, s, u, l, vt, k); //todo S vs A
        checkStatus(status);

		dim3 grid(k / BLOCK_ROWS, k / BLOCK_ROWS, 1);
		dim3 threads(BLOCK_ROWS, BLOCK_ROWS, 1);
		transpose<<<grid, threads>>>(v, vt, k, k);

		//Init Xi
		err = hipMemset(xi, 0, l*k*sizeof(float));
		checkCudaError(err);

		for (int ii=0; ii < p; ++ii){
            int i = pp[ii];

            err = hipMemcpy(ui, u + i*l, l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
						
			err = hipMemcpy(vi, vt + i*k, k * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
			
            stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, l, k, 1, &s[i], ui, l, vi, k, 0, xii, l); //todo v vs vt
            checkCublasStatus(stat);

			int threadsPerBlock = 256;
			int blocksPerGrid = (l*k + threadsPerBlock - 1) / threadsPerBlock;
			vector_add<<<blocksPerGrid, threadsPerBlock>>>(xi, xii, xi, l*k);
        }

		//Calculate Pr matrix
		err = hipMemset(v2, 0, sizeof(float));
		checkCudaError(err);

		err = hipMemset(r, 0, ld*sizeof(float));
		checkCudaError(err);

		for (int i=0; i < m; ++i){
			err = hipMemcpy(vd + i*ld, u + i*l, ld * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
		}

		dim3 grid(m / BLOCK_ROWS, ld / BLOCK_ROWS, 1);
		dim3 threads(BLOCK_ROWS, BLOCK_ROWS, 1);
		calc_r<<<grid, threads>>>(pi, u, r, vd, v2, l, m); //todo

		for (int j=0; j < ld; ++j){
            r[j] /= (1-v2);
        }

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ld, ld, m, 1, vd, ld, vd, ld, 0, vdxvdt, ld);
		checkCublasStatus(stat);

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ld, ld, 1, 1-v2, r, ld, r, ld, 0, rxrt, ld);
		checkCublasStatus(stat);

		int threadsPerBlock = 256;
		int blocksPerGrid = (l*k + threadsPerBlock - 1) / threadsPerBlock;
		vector_add<<<blocksPerGrid, threadsPerBlock>>>(vdxvdt, rxrt, pr, ld*ld);


		
		















		hipblasDestroy(handle);
		culaShutdown();
	}


}

