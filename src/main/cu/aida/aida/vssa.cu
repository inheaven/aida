#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <culapackdevice.h>
#include <culablasdevice.h>

void checkStatus(culaStatus status)
{
    char buf[80];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    //culaShutdown();
    //exit(EXIT_FAILURE);
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    //culaShutdown();
    //exit(EXIT_FAILURE);
}

void checkCublasStatus(hipblasStatus_t stat)
{
	if (stat != HIPBLAS_STATUS_SUCCESS ) {
		printf ( "CUBLAS Error \n" );

		//culaShutdown();
		//exit(EXIT_FAILURE);
	}	    
}

#define TILE_DIM    16
#define BLOCK_ROWS  16
#define BLOCK_DIM 16


// Функция транспонирования матрицы c использования разделяемой памяти
//
// inputMatrix - указатель на исходную матрицу
// outputMatrix - указатель на матрицу результат
// width - ширина исходной матрицы (она же высота матрицы-результата)
// height - высота исходной матрицы (она же ширина матрицы-результата)
//
__global__ void transposeMatrixFast(float* inputMatrix, float* outputMatrix, int width, int height)
{
    __shared__ float temp[BLOCK_DIM][BLOCK_DIM];

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xIndex < width) && (yIndex < height))
    {
        // Линейный индекс элемента строки исходной матрицы
        int idx = yIndex * width + xIndex;

        //Копируем элементы исходной матрицы
        temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
    }

    //Синхронизируем все нити в блоке
    __syncthreads();

    xIndex = blockIdx.y * blockDim.y + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;

    if ((xIndex < height) && (yIndex < width))
    {
        // Линейный индекс элемента строки исходной матрицы
        int idx = yIndex * height + xIndex;

        //Копируем элементы исходной матрицы
         outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
    }
}

__global__ void vector_add(const float *A, const float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
	
    if (i < N){
        C[i] = A[i] + B[i];		 
    }
}

//pi[i] = u[l-1 + i*l];
__global__ void calc_pi(float *u, float *v2, int l, int m){
	float v2_local = 0;

	if (blockIdx.x == 0 && threadIdx.x == 0){
		for (int i = 0; i < m; i++){			
			v2_local += u[l-1 + i*l] * u[l-1 + i*l];
		}

		v2[0] = v2_local;		
	}
}

__global__ void calc_r(float *vd, float *u, float *r, float *ra, float *v2, int ld, int m){
    int j = blockDim.x * blockIdx.x + threadIdx.x;

	float r_local = 0;

	if (j < ld){
		for (int i = 0; i < m; ++i){        
            r_local += vd[j + i*ld] * u[ld + i*(ld+1)];
        }

		r[j] = r_local;
		ra[j] = r_local/(1-v2[0]);
    }
}

__global__ void calc_zi(float *zi, float *r, float *yd, int ld){
	if (blockIdx.x == 0 && threadIdx.x == 0){
		float zi_local = 0;

		for (int i = 0; i < ld; ++i){
			zi_local += r[i] * yd[i];
		}

		zi[ld] = zi_local;
	}
}

__device__ float getSum(float *y, int rows, int cols, int first, int last, int k){
    float sum = 0;

    for (int m = first; m <= last; ++m){
        sum += rows < cols ? y[m - 1 + (k - m + 1)*rows] : y[k - m + 1 + (m - 1)*rows];
    }

    return sum;
}

__global__ void diagonalAveraging(float *y, int rows, int cols, float *g, int begin){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int l1 = rows;
    int k1 = cols;
    int n = l1 + k1 - 1;

    if (i < l1 - 1){
        g[begin + i] = getSum(y, rows, cols, 1, i + 1, i) / (i + 1);
    }else if (i >= l1 - 1 && i < k1){
        g[begin + i] = getSum(y, rows, cols, 1, l1, i) / l1;
    }else if (i >= k1 && i < n){
        g[begin + i] = getSum(y, rows, cols, i - k1 + 2, n - k1 + 1, i) / (n - i);
    }
}

void printArray(float *a, int size){
	printf("\n");

	for (int i=0; i < size; ++i){
		printf("%10.4f",  a[i]);
		
		if ((i+1)%16==0) printf("\n");
	}

	printf("\n");
}

void d_printArray(float *d_a, int size){
	float *a = (float *)malloc(size * sizeof(float));

	hipMemcpy(a, d_a, size * sizeof(float), hipMemcpyDeviceToHost);

	printArray(a, size);
}

extern "C" __declspec(dllexport) void vssa(int n, int l, int p, int* pp, int m, float* timeseries, float* forecast, int count){
	hipError_t err;
    culaStatus status;
	hipblasStatus_t stat ;
	hipblasHandle_t handle ;

	int f_size = n + m + l - 1; 
	int k = n - l + 1;
	int ld = l - 1;
	
	//Init and copy timeseries to device
	float* d_timeseries;

	err = hipMalloc(&d_timeseries, (n+count)*sizeof(float));
	checkCudaError(err);

	err = hipMemcpy(d_timeseries, timeseries, (n+count-1)*sizeof(float), hipMemcpyHostToDevice);
	checkCudaError(err);
	
	//Init and copy forecast 
	float* d_forecast;
	
	err = hipMalloc(&d_forecast, f_size*count*sizeof(float));
	checkCudaError(err);

	//Init Cublas
	stat = hipblasCreate(&handle);
	checkCublasStatus(stat);

	//Init CULA
	status = culaInitialize();
    checkStatus(status);
	
	//Init local variable
	float* x;
	err = hipMalloc(&x, l*k*sizeof(float));
	checkCudaError(err);

	float *s;
	err = hipMalloc(&s, l*sizeof(float));
	checkCudaError(err);

	float *s_h = (float *)malloc(l*sizeof(float));	

	float *u;
	err = hipMalloc(&u, l*l*sizeof(float));
	checkCudaError(err);

	float *vt;
	err = hipMalloc(&vt, k*k*sizeof(float));
	checkCudaError(err);

	float *v;
	err = hipMalloc(&v, k*k*sizeof(float));
	checkCudaError(err);

	float *xi;
	err = hipMalloc(&xi, l*k*sizeof(float));
	checkCudaError(err);

	float *ui;
	err = hipMalloc(&ui, l*sizeof(float));
	checkCudaError(err);

	float *vi;
	err = hipMalloc(&vi, k*sizeof(float));
	checkCudaError(err);

	float *xii;
	err = hipMalloc(&xii, l*k*sizeof(float));
	checkCudaError(err);

	float *pi;
	err = hipMalloc(&pi, m*sizeof(float));
	checkCudaError(err);

	float *vd;
	err = hipMalloc(&vd, ld*m*sizeof(float));
	checkCudaError(err);

	float *v2;
	err = hipMalloc(&v2, sizeof(float));
	checkCudaError(err);

	float *r;
	err = hipMalloc(&r, ld*sizeof(float));
	checkCudaError(err);

	float *ra;
	err = hipMalloc(&ra, ld*sizeof(float));
	checkCudaError(err);

	float *vdxvdt;
	err = hipMalloc(&vdxvdt, ld*ld*sizeof(float));
	checkCudaError(err);

	float *rxrt;
	err = hipMalloc(&rxrt, ld*ld*sizeof(float));
	checkCudaError(err);
	
	float *pr;
	err = hipMalloc(&pr, ld*ld*sizeof(float));
	checkCudaError(err);

	float *z;
	err = hipMalloc(&z, l*(n+m)*sizeof(float));
	checkCudaError(err);

	float *zi;
	err = hipMalloc(&zi, l*sizeof(float));
	checkCudaError(err);

	float *yd;
	err = hipMalloc(&yd, ld*sizeof(float));
	checkCudaError(err);
	
	int threads_x = BLOCK_ROWS*BLOCK_ROWS;
		
	int grid_m = (m + threads_x - 1)/threads_x;
	int grid_ld = (ld + threads_x - 1)/threads_x;
	int grid_ld2 = (ld*ld + threads_x - 1)/threads_x;
	int grid_l_k = (l*k + threads_x - 1)/threads_x;
	int grid_f_size = (f_size + threads_x - 1)/threads_x;

	dim3 grid_k_k((k + BLOCK_ROWS - 1) / BLOCK_ROWS, (k + BLOCK_ROWS - 1) / BLOCK_ROWS, 1);		
	dim3 grid_m_ld((m + BLOCK_ROWS - 1)/BLOCK_ROWS, (ld + BLOCK_ROWS - 1)/BLOCK_ROWS);
    dim3 threads_x_y(BLOCK_ROWS, BLOCK_ROWS);

	//Execute vssa for count points
	for (int index = 0; index < count; ++index){
		//Populate trajectory matrix
		for (int j = 0; j < k; ++j){
			err = hipMemcpy(x + j*l, d_timeseries + (j + index), l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
        }	
		
		//Execute SVD
		status = culaDeviceSgesvd('S', 'S', l, k, x, l, s, u, l, vt, k); //todo S vs A
        checkStatus(status);

		//copy s to host
		err = hipMemcpy(s_h, s, l * sizeof(float), hipMemcpyDeviceToHost);
		checkStatus(status);		
						
		transposeMatrixFast<<<grid_k_k, threads_x_y>>>(vt, v, k, k);
		hipDeviceSynchronize();
				
		//Init Xi
		err = hipMemset(xi, 0, l*k*sizeof(float));
		checkCudaError(err);
		
		//Alpha and beta const
		const float alpha_one = 1.0f;
		const float beta_zero = 0.0f;	
		
		for (int ii=0; ii < p; ++ii){
            int i = pp[ii];

            err = hipMemcpy(ui, u + i*l, l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
						
			err = hipMemcpy(vi, v + i*k, k * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
												
            stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, l, k, 1, &s_h[i], ui, l, vi, k, &beta_zero, xii, l); //todo v vs vt			
            checkCublasStatus(stat);			
			
			vector_add<<<grid_l_k, threads_x>>>(xi, xii, xi, l*k);
			hipDeviceSynchronize();						
        }		

		//Calculate Pr matrix
		for (int i=0; i < m; ++i){
			err = hipMemcpy(vd + i*ld, u + i*l, ld * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
		}
		
		calc_pi<<<grid_m, threads_x>>>(u, v2, l, m);
		hipDeviceSynchronize();
		
		calc_r<<<grid_ld, threads_x>>>(vd, u, r, ra, v2, ld, m);
		hipDeviceSynchronize();
				
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ld, ld, m, &alpha_one, vd, ld, vd, ld, &beta_zero, vdxvdt, ld);
		checkCublasStatus(stat);

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ld, ld, 1, &alpha_one, ra, ld, r, ld, &beta_zero, rxrt, ld);
		checkCublasStatus(stat);

		vector_add<<<grid_ld2, threads_x>>>(vdxvdt, rxrt, pr, ld*ld);
		hipDeviceSynchronize();
		
		//Calculate Z
		err = hipMemcpy(z, xi, l*k*sizeof(float), hipMemcpyDeviceToDevice);
		checkCudaError(err);
								
		for (int i = k; i < n + m; ++i){
		    err = hipMemcpy(yd, z + (1 + (i-1)*l), ld * sizeof(float), hipMemcpyDeviceToDevice);
		    checkCudaError(err);
			
		    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ld, 1, ld, &alpha_one, pr, ld, yd, ld, &beta_zero, zi, ld);
		    checkCublasStatus(stat);						
			
		    calc_zi<<<1, 1>>>(zi, ra, yd, ld);
			hipDeviceSynchronize();
			
		    err = hipMemcpy(z + i*l, zi, l * sizeof(float), hipMemcpyDeviceToDevice);
		    checkCudaError(err);
		}
		
		diagonalAveraging<<<grid_f_size, threads_x>>>(z, l, n + m, d_forecast, f_size*index);
	}

    err = hipMemcpy(forecast, d_forecast, f_size*count*sizeof(float), hipMemcpyDeviceToHost);
    checkCudaError(err);

    hipFree(d_timeseries);
    hipFree(d_forecast);
    hipFree(x);
    hipFree(s);
    hipFree(u);
    hipFree(vt);
    hipFree(v);
    hipFree(xi);
    hipFree(ui);
    hipFree(vi);
    hipFree(xii);
    hipFree(pi);
    hipFree(vd);
    hipFree(v2);
    hipFree(r);
    hipFree(ra);
    hipFree(vdxvdt);
    hipFree(rxrt);
    hipFree(pr);
    hipFree(z);
    hipFree(zi);
    hipFree(yd);

    hipblasDestroy(handle);
    culaShutdown();
}

int main(int argc, char** argv){
	printf("Hello Cuda 0!");

	float *ts = new float[32];

	for (int i = 0; i < 32; ++i){
		ts[i] = i;
	}

	float f[32+8+2-1];

	int n = 32;
	int l = 8;	
	int p = 2;
	int pp[2] = {0,1};
	int m = 2;
	
	vssa(n, l, p, pp, m, ts, f, 1);

	printArray(f, 32+8+2-1);

	printf("Hello Cuda 1!");
	return 0;
}



