#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <culapackdevice.h>

void checkStatus(culaStatus status)
{
    char buf[80];

    if(!status)
        return;

    culaGetErrorInfoString(status, culaGetErrorInfo(), buf, sizeof(buf));
    printf("%s\n", buf);

    //culaShutdown();
    //exit(EXIT_FAILURE);
}


void checkCudaError(hipError_t err)
{
    if(!err)
        return;

    printf("%s\n", hipGetErrorString(err));

    //culaShutdown();
    //exit(EXIT_FAILURE);
}

void checkCublasStatus(hipblasStatus_t stat)
{
	if (stat != HIPBLAS_STATUS_SUCCESS ) {
		printf ( "CUBLAS Error \n" );

		//culaShutdown();
		//exit(EXIT_FAILURE);
	}	    
}

#define TILE_DIM    16
#define BLOCK_ROWS  16
#define BLOCK_DIM 16


// Функция транспонирования матрицы c использования разделяемой памяти
//
// inputMatrix - указатель на исходную матрицу
// outputMatrix - указатель на матрицу результат
// width - ширина исходной матрицы (она же высота матрицы-результата)
// height - высота исходной матрицы (она же ширина матрицы-результата)
//
__global__ void transposeMatrixFast(float* inputMatrix, float* outputMatrix, int width, int height)
{
    __shared__ float temp[BLOCK_DIM][BLOCK_DIM];

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if ((xIndex < width) && (yIndex < height))
    {
        // Линейный индекс элемента строки исходной матрицы
        int idx = yIndex * width + xIndex;

        //Копируем элементы исходной матрицы
        temp[threadIdx.y][threadIdx.x] = inputMatrix[idx];
    }

    //Синхронизируем все нити в блоке
    __syncthreads();

    xIndex = blockIdx.y * blockDim.y + threadIdx.x;
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;

    if ((xIndex < height) && (yIndex < width))
    {
        // Линейный индекс элемента строки исходной матрицы
        int idx = yIndex * height + xIndex;

        //Копируем элементы исходной матрицы
         outputMatrix[idx] = temp[threadIdx.x][threadIdx.y];
    }
}

__global__ void vector_add(const float *A, const float *B, float *C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N){
        C[i] = A[i] + B[i];
    }
}

__global__ void calc_pi(float *pi, float *u, float *r, float *vd, float *v2, int l, int m){
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < m){
		pi[i] = u[l-1 + i*l];
	    v2 += pi[i] * pi[i];
	}
}

__global__ void calc_r(float *pi, float *r, float *ra, float *vd, int ld, int m){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < m && j < ld){
        ra[i] += vd[j + i*ld] * pi[i];
        r[j] += ra[i] / (1-v2); //todo +=
    }
}

__global__ void calc_zi(float *zi, float *r, float *yd, int ld){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < ld){
        zi[ld] += r[j] * yd[j];
    }
}

__device__ float getSum(float *y, int rows, int cols, int first, int last, int k){
    float sum = 0;

    for (int m = first; m <= last; ++m){
        sum += rows < cols ? y[m - 1 + (k - m + 1)*rows] : y[k - m + 1 + (m - 1)*rows];
    }

    return sum;
}

__global__ void diagonalAveraging(float *z, int rows, int cols, float *g, int begin_pos){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    int l1 = l;
    int k1 = cols;
    int n = l1 + k1 - 1;

    if (k < l1 - 1){
        g[begin + k] = getSum(y, rows, cols, 1, k + 1, k) / (k + 1);
    }else if (k >= l1 - 1 && k < k1){
        g[begin + k] = getSum(y, rows, cols, 1, l1, k) / l1;
    }else if (k >= k1 && k < n){
        g[begin + k] = getSum(y, rows, cols, k - k1 + 2, n - k1 + 1, k) / (n - k);
    }
}

void vssa(int n, int l, int p, int* pp, int m, float* timeseries, float* forecast, int count){
	hipError_t err;
    culaStatus status;
	hipblasStatus_t stat ;
	hipblasHandle_t handle ;

	int f_size = n + m + l - 1; 
	int k = n - l + 1;
	int ld = l - 1;
	
	//Init and copy timeseries to device
	float* d_timeseries;

	err = hipMalloc(&d_timeseries, (n+count)*sizeof(float));
	checkCudaError(err);

	err = hipMemcpy(d_timeseries, timeseries, (n+count)*sizeof(float), hipMemcpyHostToDevice);
	checkCudaError(err);

	//Init and copy forecast 
	float* d_forecast;
	
	err = hipMalloc(&d_forecast, f_size*count*sizeof(float));
	checkCudaError(err);

	//Init Cublas
	stat = hipblasCreate(&handle);
	checkCublasStatus(stat);

	//Init CULA
	status = culaInitialize();
    checkStatus(status);
	
	//Init local variable
	float* x;
	err = hipMalloc(&x, l*k*sizeof(float));
	checkCudaError(err);

	float *s;
	err = hipMalloc(&s, l*sizeof(float));
	checkCudaError(err);

	float *u;
	err = hipMalloc(&u, l*l*sizeof(float));
	checkCudaError(err);

	float *vt;
	err = hipMalloc(&vt, k*k*sizeof(float));
	checkCudaError(err);

	float *v;
	err = hipMalloc(&v, k*k*sizeof(float));
	checkCudaError(err);

	float *xi;
	err = hipMalloc(&xi, l*k*sizeof(float));
	checkCudaError(err);

	float *ui;
	err = hipMalloc(&ui, l*sizeof(float));
	checkCudaError(err);

	float *vi;
	err = hipMalloc(&vi, k*sizeof(float));
	checkCudaError(err);

	float *xii;
	err = hipMalloc(&xii, l*sizeof(float));
	checkCudaError(err);

	float *pi;
	err = hipMalloc(&pi, m*sizeof(float));
	checkCudaError(err);

	float *vd;
	err = hipMalloc(&vd, ld*m*sizeof(float));
	checkCudaError(err);

	float *v2;
	err = hipMalloc(&v2, sizeof(float));
	checkCudaError(err);

	float *r;
	err = hipMalloc(&r, ld*sizeof(float));
	checkCudaError(err);

	float *ra;
	err = hipMalloc(&ra, ld*sizeof(float));
	checkCudaError(err);

	float *vdxvdt;
	err = hipMalloc(&vdxvdt, ld*ld*sizeof(float));
	checkCudaError(err);

	float *rxrt;
	err = hipMalloc(&rxrt, ld*ld*sizeof(float));
	checkCudaError(err);
	
	float *pr;
	err = hipMalloc(&pr, ld*ld*sizeof(float));
	checkCudaError(err);

	float *z;
	err = hipMalloc(&z, l*(n+m)*sizeof(float));
	checkCudaError(err);

	float *zi;
	err = hipMalloc(&zi, l*sizeof(float));
	checkCudaError(err);

	float *yd;
	err = hipMalloc(&yd, ld*sizeof(float));
	checkCudaError(err);

	int threads_x = BLOCK_ROWS*BLOCK_ROWS;
	int grid_m = m/threads_x;
	int grid_ld = ld/threads_x;
	int grid_ld2 = ld*ld/threads_x;
	int grid_lk = l*k/threads_x;
	int grid_f_size = f_size/threads_x;
	dim3 grid_m_ld(m/BLOCK_ROWS, ld/BLOCK_ROWS);
    dim3 threads_x_y(BLOCK_ROWS, BLOCK_ROWS);

	//Execute vssa for count points
	for (int index = 0; index < count; ++index){
		//Populate trajectory matrix
		for (int j = 0; j < k; ++j){
			err = hipMemcpy(x + j*l, d_timeseries + (j + index), l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
        }

		//Execute SVD
		status = culaDeviceSgesvd('S', 'S', l, k, x, l, s, u, l, vt, k); //todo S vs A
        checkStatus(status);

		dim3 grid(k / BLOCK_ROWS, k / BLOCK_ROWS, 1);
		dim3 threads(BLOCK_ROWS, BLOCK_ROWS, 1);
		transposeMatrixFast<<<grid, threads>>>(v, vt, k, k);

		//Init Xi
		err = hipMemset(xi, 0, l*k*sizeof(float));
		checkCudaError(err);

		for (int ii=0; ii < p; ++ii){
            int i = pp[ii];

            err = hipMemcpy(ui, u + i*l, l * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
						
			err = hipMemcpy(vi, vt + i*k, k * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
			
            stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, l, k, 1, &s[i], ui, l, vi, k, 0, xii, l); //todo v vs vt
            checkCublasStatus(stat);

			vector_add<<<grid_l_k, threads_x>>>(xi, xii, xi, l*k); //todo xi+=
        }

		//Calculate Pr matrix
		err = hipMemset(v2, 0, sizeof(float));
		checkCudaError(err);

		err = hipMemset(r, 0, ld*sizeof(float));
		checkCudaError(err);

		for (int i=0; i < m; ++i){
			err = hipMemcpy(vd + i*ld, u + i*l, ld * sizeof(float), hipMemcpyDeviceToDevice);
			checkCudaError(err);
		}

		calc_pi<<<grid_m, threads_x>>>(pi, u, r, vd, v2, l, m);

		calc_r<<<grid_m_ld, threads_x_y>>>(pi, r, ra, vd, v2, ld, m);

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, ld, ld, m, 1, vd, ld, vd, ld, 0, vdxvdt, ld);
		checkCublasStatus(stat);

        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ld, ld, 1, 1, ra, ld, r, ld, 0, rxrt, ld);
		checkCublasStatus(stat);

		vector_add<<<grid_ld2, threads_x>>>(vdxvdt, rxrt, pr, ld*ld);

		//Calculate Z
		err = cudaMemcpu(z, xi, (long)l*k*sizeof(float), hipMemcpyDeviceToDevice);
		checkCudaError(err);

		for (int i = k; i < n + m; ++i){
		    err = hipMemcpy(yd, z + 1 + (i-1)*l, ld * sizeof(float), hipMemcpyDeviceToDevice);
		    checkCudaError(err);

		    err = hipMemset(zi, 0, l*sizeof(float));
		    checkCudaError(err);

		    stat = hipblasSgemm(HIPBLAS_OP_N, HIPBLAS_OP_N, ld, 1, ld, 1, pr, ld, yd, ld, 0, zi, ld);
		    checkCublasStatus(stat);

		    calc_zi<<<grid_ld, threads_x>>>(zi, r, yd, ld);

		    err = hipMemcpy(z + i*l, zi, l * sizeof(float), hipMemcpyDeviceToDevice);
		    checkCudaError(err);
		}

		diagonalAveraging<<<grid_f_size, threads_x>>>(z, l, n + m, d_forecast, f_size*index);
	}

    err = hipMemcpy(d_forecast, forecast, f_size*count*sizeof(float), hipMemcpyDeviceToHost);
    checkCudaError(err);

    hipFree(d_timeseries);
    hipFree(d_forecast);
    hipFree(x);
    hipFree(s);
    hipFree(u);
    hipFree(vt);
    hipFree(v);
    hipFree(xi);
    hipFree(ui);
    hipFree(vi);
    hipFree(xii);
    hipFree(pi);
    hipFree(vd);
    hipFree(v2);
    hipFree(r);
    hipFree(ra);
    hipFree(vdxvdt);
    hipFree(rxrt);
    hipFree(pr);
    hipFree(z);
    hipFree(zi);
    hipFree(yd);

    hipblasDestroy(handle);
    culaShutdown();
}



